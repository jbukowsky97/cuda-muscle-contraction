
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 128

__global__ void calculateWork(int* work, const unsigned long long int leftMiddle, const unsigned long long int middle, const unsigned long long int n) {
    int i = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    int temp = i % n;
    int force;
    if (temp < leftMiddle) {
        force = middle - (leftMiddle - temp);
    } else if (temp == leftMiddle || temp == middle) {
        force = middle;
    } else {
        force = n - temp;
    }
    int distance = i % 10 + 1;

    work[i] = force * distance;
}

extern "C" void gpuCalculate(int* w, const unsigned long long int leftMiddle, const unsigned long long int middle, const unsigned long long int n) {
    hipError_t mallocResult;
    int* work;

    mallocResult = hipMalloc((void**) &work, n * sizeof(int));
    if (mallocResult != hipSuccess) {
        fprintf(stderr, "CUDA Malloc failed, exiting...\n");
        exit(EXIT_FAILURE);
    }

    dim3 dimBlock(BLOCK_SIZE);
    unsigned long long int gridSize = n / BLOCK_SIZE;
    if (n % BLOCK_SIZE != 0) {
        gridSize += 1;
    }
    dim3 dimGrid(gridSize);

    calculateWork<<<dimGrid, dimBlock>>>(work, leftMiddle, middle, n);

    mallocResult = hipMemcpy(w, work, n * sizeof(int), hipMemcpyDeviceToHost);
    if (mallocResult != hipSuccess) {
        fprintf(stderr, "CUDA Memcpy failed, exiting...\n");
        exit(EXIT_FAILURE);
    }

    mallocResult = hipFree(work);
    if (mallocResult != hipSuccess) {
        fprintf(stderr, "CUDA free failed, exiting...\n");
        exit(EXIT_FAILURE);
    }
}